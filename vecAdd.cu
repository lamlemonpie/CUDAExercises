#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>


__global__ 
void vecAddKernel(float *d_vec1, float *d_vec2, float *d_out, int n) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < n)
    d_out[i] = d_vec1[i] + d_vec2[i];
  }


void vecAdd(float *h_vec1, float *h_vec2, float *h_out, int n){

  int size = n*sizeof(float);   //Cantidad de memoria.
  float *d_vec1, *d_vec2, *d_out;

  hipMalloc( (void**)&d_vec1 ,size);
  hipMalloc( (void**)&d_vec2 ,size);
  hipMalloc( (void**)&d_out  ,size);

  hipMemcpy(d_vec1, h_vec1, size, hipMemcpyHostToDevice);
  hipMemcpy(d_vec2, h_vec2, size, hipMemcpyHostToDevice);


  vecAddKernel<<<ceil(n/256.0),256>>>(d_vec1,d_vec2,d_out,n);

  hipMemcpy(h_out, d_out, size, hipMemcpyDeviceToHost);

  hipFree(d_vec1);
  hipFree(d_vec2);
  hipFree(d_out);

}

void genVector(float *x, int n) {
  for (int i = 0; i < n; i++)
     x[i] = random()/((float) RAND_MAX);
}

void printVector(const char* title, float *y, double n) {
  printf("%s\n", title);
  for (int i = 0; i < n; i++)
     printf("%4.1f ", y[i]);
  printf("\n");
} 

int main(int argc, char **argv){
  
  int n = 30;
  float *h_vec1 = NULL;
  float *h_vec2 = NULL;
  float *h_out  = NULL;

  h_vec1 = (float *) malloc( n*sizeof(float) );
  h_vec2 = (float *) malloc( n*sizeof(float) );
  h_out  = (float *) malloc( n*sizeof(float) );
  
  const char * v1 = "Vector1";
  const	char * v2 = "Vector2";
  const	char * out = "Salida";

  genVector(h_vec1,n); genVector(h_vec2,n);
  printVector(v1,h_vec1,n);
  printVector(v2,h_vec2,n);

  vecAdd(h_vec1,h_vec2,h_out,n);

  printVector(out,h_out,n);
  
  return 0;
}
